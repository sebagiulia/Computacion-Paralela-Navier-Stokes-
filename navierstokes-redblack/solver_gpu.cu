#include "hip/hip_runtime.h"
#include <stddef.h>
#include <hip/hip_runtime.h>

#include "solver.h"

#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}

typedef enum { NONE = 0, VERTICAL = 1, HORIZONTAL = 2 } boundary;
typedef enum { RED, BLACK } grid_color;

#define IX(x,y) (rb_idx((x),(y),(n+2)))

__device__ static inline size_t rb_idx(size_t x, size_t y, size_t dim) {
    size_t base = ((x % 2) ^ (y % 2)) * dim * (dim / 2);
    size_t offset = (x / 2) + y * (dim / 2);
    return base + offset;
}

__global__ static void add_source(unsigned int n, float * x, const float * s, float dt)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x; 
    x[i] += dt * s[i];
}


__device__ static void set_bnd(unsigned int n, boundary b, float * x)
{
    for (unsigned int i = 1; i <= n; i++) {
        x[IX(0, i)]     = b == VERTICAL ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(n + 1, i)] = b == VERTICAL ? -x[IX(n, i)] : x[IX(n, i)];
        x[IX(i, 0)]     = b == HORIZONTAL ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, n + 1)] = b == HORIZONTAL ? -x[IX(i, n)] : x[IX(i, n)];
    }
    x[IX(0, 0)]         = 0.5f * (x[IX(1, 0)]     + x[IX(0, 1)]);
    x[IX(0, n + 1)]     = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
    x[IX(n + 1, 0)]     = 0.5f * (x[IX(n, 0)]     + x[IX(n + 1, 1)]);
    x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

__device__ static void lin_solve_rb_step(grid_color color,
                              unsigned int n,
                              float a,
                              float c,
                              const float * __restrict__ same0,
                              const float * __restrict__ neigh,
                              float * __restrict__ same)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
    
    int shift_color = color == RED ? 1 : -1;
    int shift_gid   = gid % 2 == 0 ? 1 : -1;
    int shift       = shift_color * shift_gid;

    unsigned int start = color == RED ? gid % 2 : (1 - (gid % 2));
    unsigned int width = (n + 2) / 2;
    unsigned int index = gid + width;
    //for (unsigned int y = 1; y <= n; ++y, shift = -shift, start = 1 - start) {
        //for (unsigned int x = start; x < width - (1 - start); ++x) {
            same[index] = (same0[index] + a * (neigh[index - width] +
                                               neigh[index] +
                                               neigh[index + shift] +
                                               neigh[index + width])) / c;
        //}
    //}
}

__global__ static void lin_solve(unsigned int n, boundary b,
                      float * __restrict__ x,
                      const float * __restrict__ x0,
                      float a, float c)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
    
    unsigned int color_size = (n + 2) * ((n + 2) / 2);
    const float * red0 = x0;
    const float * blk0 = x0 + color_size;
    float * red = x;
    float * blk = x + color_size;
    for (unsigned int k = 0; k < 20; ++k) {
        lin_solve_rb_step(RED,   n, a, c, red0, blk, red);
        __syncthreads();
        lin_solve_rb_step(BLACK, n, a, c, blk0, red, blk);
	
	if (gid == 0)
	    set_bnd(n, b, x);
	
	__syncthreads();
    }
}


static void diffuse(unsigned int n, boundary b, float * x, const float * x0, float diff, float dt)
{
	float a = dt * diff * n * n;
	unsigned int block_size = 128;
	unsigned int active_cells = n * (n / 2);
	unsigned int num_blocks = (active_cells + block_size - 1) / block_size;

	lin_solve<<<num_blocks, block_size>>>(n, b, x, x0, a, 1 + 4 * a);
}

__global__ static void advect_kernel(unsigned int n, boundary b, float * d, const float * d0, const float * u, const float * v, float dt)
{
    int i0, i1, j0, j1, i, j;
    float x, y, s0, t0, s1, t1;
    float dt0 = dt * n;
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
    i = (gid / n) + 1; 
    j = (gid % n) + 1;
    x = i - dt0 * u[IX(i,j)];
    y = j - dt0 * v[IX(i,j)];
    if (x < 0.5f) {
	x = 0.5f;
    } else if (x > n + 0.5f) {
	x = n + 0.5f;
    }
    i0 = (int) x;
    i1 = i0 + 1;
    if (y < 0.5f) {
	y = 0.5f;
    } else if (y > n + 0.5f) {
	y = n + 0.5f;
    }
    j0 = (int) y;
    j1 = j0 + 1;
    s1 = x - i0;
    s0 = 1 - s1;
    t1 = y - j0;
    t0 = 1 - t1;
    d[IX(i,j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
		 s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    __syncthreads();
    if (gid == 0)
    	set_bnd(n, b, d);
}

static void advect(unsigned int n, boundary b, float * d, const float * d0, const float * u, const float * v, float dt)
{
    unsigned int block_size = 128;
    unsigned int num_blocks = n * n / block_size;
    advect_kernel<<<num_blocks,block_size>>>(n,b,d,d0,u,v,dt);
}

__global__ static void project_kernel1(unsigned int n, float *u, float *v, float *p, float *div)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int i = (gid / n) + 1; 
    unsigned int j = (gid % n) + 1;
    div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
			     v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
    p[IX(i, j)] = 0;
    __syncthreads();
    if (gid == 0)
    {
        set_bnd(n, NONE, div);
        set_bnd(n, NONE, p);
    }
}


__global__ static void project_kernel2(unsigned int n, float *u, float *v, float *p, float *div)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int i = (gid / n) + 1; 
    unsigned int j = (gid % n) + 1;
    u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
    __syncthreads();
    if (gid == 0)
    {
	set_bnd(n, VERTICAL, u);
	set_bnd(n, HORIZONTAL, v);
    }
}

static void project(unsigned int n, float *u, float *v, float *p, float *div)
{
    unsigned int block_size = 128;
    unsigned int num_blocks = (n + 2) * ((n + 2) / 2) / block_size;
    unsigned int num_blocks_p = n * n / block_size;
    project_kernel1<<<num_blocks_p,block_size>>>(n, u, v, p, div);
    lin_solve<<<num_blocks,block_size>>>(n, NONE, p, div, 1, 4);
    project_kernel2<<<num_blocks_p,block_size>>>(n, u, v, p, div);
}

void dens_step(unsigned int n, float *x, float *x0, float *u, float *v, float diff, float dt)
{
    unsigned int block_size = 128;
    unsigned int num_blocks = n * n / block_size;
    add_source<<<num_blocks,block_size>>>(n, x, x0, dt);
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

void vel_step(unsigned int n, float *u, float *v, float *u0, float *v0, float visc, float dt)
{
    unsigned int block_size = 128;
    unsigned int num_blocks = n * n / block_size;
    add_source<<<num_blocks,block_size>>>(n, u, u0, dt);
    add_source<<<num_blocks,block_size>>>(n, v, v0, dt);
    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
