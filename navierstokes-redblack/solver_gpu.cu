#include "hip/hip_runtime.h"
#include <stddef.h>
#include <hip/hip_runtime.h>

#include "solver.h"
#include "indices.h"

#define IX(x,y) (rb_idx((x),(y),(n+2)))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}

typedef enum { NONE = 0, VERTICAL = 1, HORIZONTAL = 2 } boundary;
typedef enum { RED, BLACK } grid_color;

static void add_source(unsigned int n, float * x, const float * s, float dt)
{
    unsigned int size = (n + 2) * (n + 2);
    for (unsigned int i = 0; i < size; i++) {
        x[i] += dt * s[i];
    }
}

static void set_bnd(unsigned int n, boundary b, float * x)
{
    for (unsigned int i = 1; i <= n; i++) {
        x[IX(0, i)]     = b == VERTICAL ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(n + 1, i)] = b == VERTICAL ? -x[IX(n, i)] : x[IX(n, i)];
        x[IX(i, 0)]     = b == HORIZONTAL ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, n + 1)] = b == HORIZONTAL ? -x[IX(i, n)] : x[IX(i, n)];
    }
    x[IX(0, 0)]         = 0.5f * (x[IX(1, 0)]     + x[IX(0, 1)]);
    x[IX(0, n + 1)]     = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
    x[IX(n + 1, 0)]     = 0.5f * (x[IX(n, 0)]     + x[IX(n + 1, 1)]);
    x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

__global__ static void lin_solve_rb_step(grid_color color,
                              unsigned int n,
                              float a,
                              float c,
                              const float * __restrict__ same0,
                              const float * __restrict__ neigh,
                              float * __restrict__ same)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
    if (gid >= n)
	    return;
    
    int shift_color = color == RED ? 1 : -1;
    int shift_gid   = gid % 2 == 0 ? 1 : -1;
    int shift       = shift_color * shift_gid;

    unsigned int start = color == RED ? gid % 2 : (1 - (gid % 2));
    /* R N R N R N    -> SHIFT =  1, START = 0, GID = 0
     * N R N R N R    -> SHIFT = -1, START = 1, GID = 1
     * R N R N R N    -> SHIFT =  1, START = 0, GID = 2
     * N R N R N R    -> SHIFT = -1, START = 1, GID = 3
     *
     *
     *
     */

    unsigned int width = (n + 2) / 2;
    unsigned int y = gid + 1;
    //for (unsigned int y = 1; y <= n; ++y, shift = -shift, start = 1 - start) {
        for (unsigned int x = start; x < width - (1 - start); ++x) {
            int index = idx(x, y, width);
            same[index] = (same0[index] + a * (neigh[index - width] +
                                               neigh[index] +
                                               neigh[index + shift] +
                                               neigh[index + width])) / c;
        }
    //}
}

__global__ static void lin_solve(unsigned int n, boundary b,
                      float * __restrict__ x,
                      const float * __restrict__ x0,
                      float a, float c)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
    size_t tid = threadIdx.x;
    size_t lid = tid % warpSize;
    
    unsigned int color_size = (n + 2) * ((n + 2) / 2);
    const float * red0 = x0;
    const float * blk0 = x0 + color_size;
    float * red = x;
    float * blk = x + color_size;
    for (unsigned int k = 0; k < 20; ++k) {
        lin_solve_rb_step(RED,   n, a, c, red0, blk, red);
        __synchthreads();
        lin_solve_rb_step(BLACK, n, a, c, blk0, red, blk);
        __synchthreads();
	set_bnd(n, b, x);
	__synchthreads();

    }
}

static void diffuse(unsigned int n, boundary b, float * x, const float * x0, float diff, float dt)
{
    float a = dt * diff * n * n;
    lin_solve(n, b, x, x0, a, 1 + 4 * a);
}

static void advect(unsigned int n, boundary b, float * d, const float * d0, const float * u, const float * v, float dt)
{
    int i0, i1, j0, j1;
    float x, y, s0, t0, s1, t1;

    float dt0 = dt * n;
    for (unsigned int i = 1; i <= n; i++) {
        for (unsigned int j = 1; j <= n; j++) {
            x = i - dt0 * u[IX(i, j)];
            y = j - dt0 * v[IX(i, j)];
            if (x < 0.5f) {
                x = 0.5f;
            } else if (x > n + 0.5f) {
                x = n + 0.5f;
            }
            i0 = (int) x;
            i1 = i0 + 1;
            if (y < 0.5f) {
                y = 0.5f;
            } else if (y > n + 0.5f) {
                y = n + 0.5f;
            }
            j0 = (int) y;
            j1 = j0 + 1;
            s1 = x - i0;
            s0 = 1 - s1;
            t1 = y - j0;
            t0 = 1 - t1;
            d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
                          s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
        }
    }
    set_bnd(n, b, d);
}

static void project(unsigned int n, float *u, float *v, float *p, float *div)
{
    for (unsigned int i = 1; i <= n; i++) {
        for (unsigned int j = 1; j <= n; j++) {
            div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
                                     v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
            p[IX(i, j)] = 0;
        }
    }
    set_bnd(n, NONE, div);
    set_bnd(n, NONE, p);

    lin_solve(n, NONE, p, div, 1, 4);

    for (unsigned int i = 1; i <= n; i++) {
        for (unsigned int j = 1; j <= n; j++) {
            u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
            v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
        }
    }
    set_bnd(n, VERTICAL, u);
    set_bnd(n, HORIZONTAL, v);
}

void dens_step(unsigned int n, float *x, float *x0, float *u, float *v, float diff, float dt)
{
    add_source(n, x, x0, dt);
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

void vel_step(unsigned int n, float *u, float *v, float *u0, float *v0, float visc, float dt)
{
    add_source(n, u, u0, dt);
    add_source(n, v, v0, dt);
    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
