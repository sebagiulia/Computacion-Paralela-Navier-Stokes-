#include "hip/hip_runtime.h"
#include <stddef.h>
#include <hip/hip_runtime.h>

#include "solver.h"

#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}
#define DIV_CEIL(n, m) ((n) + (m) -1) / (m)

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 16

typedef enum { NONE = 0, VERTICAL = 1, HORIZONTAL = 2 } boundary;
typedef enum { RED, BLACK } grid_color;

#define IX(x,y) (rb_idx((x),(y),(n+2)))

__device__ static inline size_t rb_idx(size_t x, size_t y, size_t dim) {
    size_t base = ((x % 2) ^ (y % 2)) * dim * (dim / 2);
    size_t offset = (x / 2) + y * (dim / 2);
    return base + offset;
}


__global__ static void add_source_kernel(unsigned int n, float * x, const float * s, float dt)
{
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < (n+2) * (n+2)){
        x[i] += dt * s[i];
    }
}

static void add_source(uint n, float * x, const float * s, float dt)
{    
    uint block_size = 128;
    uint num_blocks = ((n + 2) * (n + 2) + block_size - 1) / block_size; 
    add_source_kernel<<<num_blocks, block_size>>>(n, x, s, dt);
    hipDeviceSynchronize();
}

__global__ static void set_bnd_kernel(uint n, boundary b, float * x)
{
    // lanzo 4 n hilos 
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint i = (idx % n) + 1, sel = idx / n;
    
    if (sel == 0) x[IX(0, i)]          = b == VERTICAL   ? -x[IX(1, i)] : x[IX(1, i)];
    else if (sel == 3) x[IX(i, n + 1)] = b == HORIZONTAL ? -x[IX(i, n)] : x[IX(i, n)];
    else if (sel == 1) x[IX(n + 1, i)] = b == VERTICAL   ? -x[IX(n, i)] : x[IX(n, i)];
    else if (sel == 2) x[IX(i, 0)]     = b == HORIZONTAL ? -x[IX(i, 1)] : x[IX(i, 1)];

    if (idx == 1)      x[IX(0, 0)]      = 0.5f * (x[IX(1, 0)]     + x[IX(0, 1)]);
    else if (idx == n) x[IX(0, n + 1)]  = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
    else if (idx == 3 * n + 1) x[IX(n + 1, 0)]     = 0.5f * (x[IX(n, 0)]     + x[IX(n + 1, 1)]);
    else if (idx == 4 * n)     x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

static void set_bnd(unsigned int n, boundary b, float * x)
{
    // dimensiones para set_bnd_kernel
    dim3 block(BLOCK_WIDTH);
    dim3 grid(DIV_CEIL(n*4, block.x));

    set_bnd_kernel<<<grid, block>>>(n, b, x);
    hipDeviceSynchronize(); // espero a que los kernels terminen
}  

__device__ static void lin_solve_rb_step(
			      uint x_aux,
			      uint y_aux,
		              grid_color color,
                              unsigned int n,
                              float a,
                              float c,
                              const float * __restrict__ same0,
                              const float * __restrict__ neigh,
                              float * __restrict__ same)
{
	    unsigned width = (n+2) / 2;
	    unsigned gid = x_aux + y_aux * width; 
	
	
    	    int shift_color = color == RED ? 1 : -1;
   	    int shift_gid   = gid % 2 == 0 ? 1 : -1;
            int shift       = shift_color * shift_gid;
   	    unsigned int start = color == RED ? gid % 2 : (1 - (gid % 2));

   	    int x = x_aux + start;
   	    int y = y_aux + 1;
	    if(x  >= width - (1 - start) || y > n)
	 	return;
  	    unsigned index = x + y * width;
    //for (unsigned int y = 1; y <= n; ++y, shift = -shift, start = 1 - start) {
        //for (unsigned int x = start; x < width - (1 - start); ++x) {
            same[index] = (same0[index] + a * (neigh[index - width] +
                                               neigh[index] +
                                               neigh[index + shift] +
                                               neigh[index + width])) / c;
        //}
    //}
}

__global__ static void lin_solve_kernel(unsigned int n,
                              float a,
                              float c,
                              const float * __restrict__ red0,
			      const float * __restrict__ blk0,
			      float * __restrict__ red,
                              float * __restrict__ blk) {

   	size_t x_aux = blockIdx.x * blockDim.x + threadIdx.x; 
   	size_t y_aux = blockIdx.y * blockDim.y + threadIdx.y; 
           
	lin_solve_rb_step(x_aux, y_aux, RED, n, a, c, red0, blk, red);
	__syncthreads();
	lin_solve_rb_step(x_aux, y_aux, BLACK, n, a, c, blk0, red, blk);
}


static void lin_solve(unsigned int n, boundary b,
                      float * __restrict__ x,
                      const float * __restrict__ x0,
                      float a, float c)
{
    int width = (n + 2) / 2;	
    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 grid(DIV_CEIL(width, block.x), DIV_CEIL(n, block.y));

    unsigned int color_size = (n + 2) * ((n + 2) / 2);
    const float * red0 = x0;
    const float * blk0 = x0 + color_size;
    float * red = x;
    float * blk = x + color_size;
    for (unsigned int k = 0; k < 20; ++k) {
        
	lin_solve_kernel<<<grid, block>>>(n, a, c, red0, blk0, red, blk);
	hipDeviceSynchronize();

	set_bnd(n, b, x);
	
    }
}




static void diffuse(unsigned int n, boundary b, float * x, const float * x0, float diff, float dt)
{

	float a = dt * diff * n * n;
	lin_solve(n, b, x, x0, a, 1 + 4 * a);
}

__global__ static void advect_kernel(unsigned int n, boundary b, float * d, const float * d0, const float * u, const float * v, float dt)
{
    int i0, i1, j0, j1, i, j;
    float x, y, s0, t0, s1, t1;
    float dt0 = dt * n;
    size_t xid = blockIdx.x * blockDim.x + threadIdx.x; 
    size_t yid = blockIdx.y * blockDim.y + threadIdx.y;

    i = xid + 1; 
    j = yid + 1;
    if(i > n || j > n) return;
    x = i - dt0 * u[IX(i,j)];
    y = j - dt0 * v[IX(i,j)];
    if (x < 0.5f) {
	x = 0.5f;
    } else if (x > n + 0.5f) {
	x = n + 0.5f;
    }
    i0 = (int) x;
    i1 = i0 + 1;
    if (y < 0.5f) {
	y = 0.5f;
    } else if (y > n + 0.5f) {
	y = n + 0.5f;
    }
    j0 = (int) y;
    j1 = j0 + 1;
    s1 = x - i0;
    s0 = 1 - s1;
    t1 = y - j0;
    t0 = 1 - t1;
    d[IX(i,j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
		 s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
}

static void advect(unsigned int n, boundary b, float * d, const float * d0, const float * u, const float * v, float dt)
{
    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 grid(DIV_CEIL(n, block.x), DIV_CEIL(n, block.y));

    advect_kernel<<<grid,block>>>(n,b,d,d0,u,v,dt);
    hipDeviceSynchronize();

    set_bnd(n, b, d);
}

__global__ static void project_kernel1(unsigned int n, float *u, float *v, float *p, float *div)
{
    size_t xid = blockIdx.x * blockDim.x + threadIdx.x; 
    size_t yid = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int i = xid + 1; 
    unsigned int j = yid + 1;
    if(i > n || j > n) return;
    div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
			     v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
    p[IX(i, j)] = 0;
}


__global__ static void project_kernel2(unsigned int n, float *u, float *v, float *p, float *div)
{
    
   size_t xid = blockIdx.x * blockDim.x + threadIdx.x; 
    size_t yid = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int i = xid + 1; 
    unsigned int j = yid + 1;
    if(i > n || j > n) return; 
    u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
}

static void project(unsigned int n, float *u, float *v, float *p, float *div)
{
    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 grid(DIV_CEIL(n, block.x), DIV_CEIL(n, block.y));

    project_kernel1<<<grid,block>>>(n, u, v, p, div);
    hipDeviceSynchronize();

    set_bnd(n, NONE, div);
    set_bnd(n, NONE, p);
    
    lin_solve(n, NONE, p, div, 1, 4);
    
    project_kernel2<<<grid,block>>>(n, u, v, p, div);
    hipDeviceSynchronize();

    set_bnd(n, VERTICAL, u);
    set_bnd(n, HORIZONTAL, v);
}

void dens_step(unsigned int n, float *x, float *x0, float *u, float *v, float diff, float dt)
{ 
    add_source(n, x, x0, dt);
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

void vel_step(unsigned int n, float *u, float *v, float *u0, float *v0, float visc, float dt)
{
    add_source(n, u, u0, dt);
    add_source(n, v, v0, dt);
    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
