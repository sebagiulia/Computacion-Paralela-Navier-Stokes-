#include "hip/hip_runtime.h"
/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

    This code is a simple prototype that demonstrates how to use the
    code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
    for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include <stdlib.h>
#include <stdio.h>
#include <GL/glut.h>
#include "indices.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "solver.h"
#include "timing.h"

/* macros */

#define IX(x,y) (rb_idx((x),(y),(N+2)))

/* global variables */

static int N;
static float dt, diff, visc;
static float force, source;
static int dvel;

static float * u, * v, * u_prev, * v_prev;
static float * dens, * dens_prev;
static float * hu, * hv, * hu_prev, * hv_prev;
static float * hdens, * hdens_prev;

static int win_id;
static int win_x, win_y;
static int mouse_down[3];
static int omx, omy, mx, my;


/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data ( void )
{
	if ( hu ) free ( hu );
	if ( hv ) free ( hv );
	if ( hu_prev ) free ( hu_prev );
	if ( hv_prev ) free ( hv_prev );
	if ( hdens ) free ( hdens );
	if ( hdens_prev ) free ( hdens_prev );
	if ( u ) hipFree ( u );
	if ( v ) hipFree ( v );
	if ( u_prev ) hipFree ( u_prev );
	if ( v_prev ) hipFree ( v_prev );
	if ( dens ) hipFree ( dens );
	if ( dens_prev ) hipFree ( dens_prev );
}

static void clear_data ( void )
{
	int i, size=(N+2)*(N+2);
	for ( i=0 ; i<size ; i++ ) {
		hu[i] = hv[i] = hu_prev[i] = hv_prev[i] = hdens[i] = hdens_prev[i] = 0.0f;
	}
}

static int allocate_data ( void )
{
	int size = (N+2)*(N+2);

	hipError_t ue = hipMalloc((float **)&u, size*sizeof(float) );
	hipError_t ve = hipMalloc((float **)&v, size*sizeof(float) );
	hipError_t dense = hipMalloc((float **)&dens, size*sizeof(float) );
	hipError_t upreve = hipMalloc((float **)&u_prev, size*sizeof(float) );
	hipError_t vpreve = hipMalloc((float **)&v_prev, size*sizeof(float) );
	hipError_t denspreve = hipMalloc((float **)&dens_prev, size*sizeof(float) );

	if (ue != hipSuccess ||
	    ve != hipSuccess ||
	    dense != hipSuccess ||
	    upreve != hipSuccess ||
	    vpreve != hipSuccess ||
	    denspreve != hipSuccess)
	{
	    fprintf ( stderr, "cannot allocate gpu data\n" );
	    return ( 0 );
	}
	hu_prev	   = (float *) malloc ( size*sizeof(float) );
	hv_prev	   = (float *) malloc ( size*sizeof(float) );
	hdens_prev = (float *) malloc ( size*sizeof(float) );
	hu	   = (float *) malloc ( size*sizeof(float) );
	hv	   = (float *) malloc ( size*sizeof(float) );
	hdens      = (float *) malloc ( size*sizeof(float) );

	if ( !hu || !hv || !hu_prev || !hv_prev || !hdens || !hdens_prev ) {
		fprintf ( stderr, "cannot allocate data\n" );
		return ( 0 );
	}

	return ( 1 );
}

__global__ void compute_velocity2(const float* u, const float* v, float* velocity2, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float ui = u[i];
        float vi = v[i];
        velocity2[i] = ui * ui + vi * vi;
    }
}


/*
  ----------------------------------------------------------------------
   OpenGL specific drawing routines
  ----------------------------------------------------------------------
*/

static void pre_display ( void )
{
	glViewport ( 0, 0, win_x, win_y );
	glMatrixMode ( GL_PROJECTION );
	glLoadIdentity ();
	gluOrtho2D ( 0.0, 1.0, 0.0, 1.0 );
	glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
}

static void post_display ( void )
{
	glutSwapBuffers ();
}

static void draw_velocity ( void )
{
	int i, j;
	float x, y, h;

	h = 1.0f/N;

	glColor3f ( 1.0f, 1.0f, 1.0f );
	glLineWidth ( 1.0f );

	glBegin ( GL_LINES );

		for ( i=1 ; i<=N ; i++ ) {
			x = (i-0.5f)*h;
			for ( j=1 ; j<=N ; j++ ) {
				y = (j-0.5f)*h;

				glVertex2f ( x, y );
				glVertex2f ( x+hu[IX(i,j)], y+hv[IX(i,j)] );
			}
		}

	glEnd ();
}

static void draw_density ( void )
{
	int i, j;
	float x, y, h, d00, d01, d10, d11;

	h = 1.0f/N;

	glBegin ( GL_QUADS );

		for ( i=0 ; i<=N ; i++ ) {
			x = (i-0.5f)*h;
			for ( j=0 ; j<=N ; j++ ) {
				y = (j-0.5f)*h;

				d00 = hdens[IX(i,j)];
				d01 = hdens[IX(i,j+1)];
				d10 = hdens[IX(i+1,j)];
				d11 = hdens[IX(i+1,j+1)];

				glColor3f ( d00, d00, d00 ); glVertex2f ( x, y );
				glColor3f ( d10, d10, d10 ); glVertex2f ( x+h, y );
				glColor3f ( d11, d11, d11 ); glVertex2f ( x+h, y+h );
				glColor3f ( d01, d01, d01 ); glVertex2f ( x, y+h );
			}
		}

	glEnd ();
}

/*
  ----------------------------------------------------------------------
   relates mouse movements to forces sources
  ----------------------------------------------------------------------
*/







static void react ( float * d_d, float * d_u, float * d_v )
{
    int size = (N + 2) * (N + 2);
    float max_velocity2, max_density;

    // --- Crear array temporal para velocity^2
    float* d_velocity2;
    hipMalloc(&d_velocity2, size * sizeof(float));

    // --- 1. Calcular u^2 + v^2
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    compute_velocity2<<<numBlocks, blockSize>>>(d_u, d_v, d_velocity2, size);

    // --- 2. Usar CUB para obtener los máximos

    // 2.a max_density
    float* d_out_density;
    void* d_temp_storage_density = nullptr;
    size_t temp_storage_bytes_density = 0;
    hipMalloc(&d_out_density, sizeof(float));
    hipcub::DeviceReduce::Max(nullptr, temp_storage_bytes_density, d_d, d_out_density, size);
    hipMalloc(&d_temp_storage_density, temp_storage_bytes_density);
    hipcub::DeviceReduce::Max(d_temp_storage_density, temp_storage_bytes_density, d_d, d_out_density, size);

    // 2.b max_velocity2
    float* d_out_velocity2;
    void* d_temp_storage_velocity = nullptr;
    size_t temp_storage_bytes_velocity = 0;
    hipMalloc(&d_out_velocity2, sizeof(float));
    hipcub::DeviceReduce::Max(nullptr, temp_storage_bytes_velocity, d_velocity2, d_out_velocity2, size);
    hipMalloc(&d_temp_storage_velocity, temp_storage_bytes_velocity);
    hipcub::DeviceReduce::Max(d_temp_storage_velocity, temp_storage_bytes_velocity, d_velocity2, d_out_velocity2, size);

    // --- 3. Copiar máximos a host
    hipMemcpy(&max_density, d_out_density, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&max_velocity2, d_out_velocity2, sizeof(float), hipMemcpyDeviceToHost);

    // --- 4. Resetear arrays a 0
    hipMemset(d_u, 0, size * sizeof(float));
    hipMemset(d_v, 0, size * sizeof(float));
    hipMemset(d_d, 0, size * sizeof(float));

    // --- 5. Aplicar condiciones si hace falta
    int center = IX(N/2, N/2); 
    if (max_velocity2 < 0.0000005f) {
        float force_val = force * 10.0f;
        hipMemcpy(&d_u[center], &force_val, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&d_v[center], &force_val, sizeof(float), hipMemcpyHostToDevice);
    }
    if (max_density < 1.0f) {
        float source_val = source * 10.0f;
        hipMemcpy(&d_d[center], &source_val, sizeof(float), hipMemcpyHostToDevice);
    }

    // --- Cleanup
    hipFree(d_velocity2);
    hipFree(d_out_density);
    hipFree(d_out_velocity2);
    hipFree(d_temp_storage_density);
    hipFree(d_temp_storage_velocity);
}


/*
  ----------------------------------------------------------------------
   GLUT callback routines
  ----------------------------------------------------------------------
*/

static void key_func ( unsigned char key, int x, int y )
{
	switch ( key )
	{
		case 'c':
		case 'C':
			clear_data ();
			break;

		case 'q':
		case 'Q':
			free_data ();
			exit ( 0 );
			break;

		case 'v':
		case 'V':
			dvel = !dvel;
			break;
	}
}

static void mouse_func ( int button, int state, int x, int y )
{
	omx = mx = x;
	omx = my = y;

	mouse_down[button] = state == GLUT_DOWN;
}

static void motion_func ( int x, int y )
{
	mx = x;
	my = y;
}

static void reshape_func ( int width, int height )
{
	glutSetWindow ( win_id );
	glutReshapeWindow ( width, height );

	win_x = width;
	win_y = height;
}

static void idle_func ( void )
{
static int size = (N+2)*(N+2);
	hipError_t err_u         = hipMemcpy(u, hu, size, hipMemcpyHostToDevice);
	hipError_t err_v         = hipMemcpy(v, hv, size, hipMemcpyHostToDevice);
	hipError_t err_dens      = hipMemcpy(dens, hdens, size, hipMemcpyHostToDevice);
	hipError_t err_u_prev    = hipMemcpy(u_prev, hu_prev, size, hipMemcpyHostToDevice);
	hipError_t err_v_prev    = hipMemcpy(v_prev, hv_prev, size, hipMemcpyHostToDevice);
	hipError_t err_dens_prev = hipMemcpy(dens_prev, hdens_prev, size, hipMemcpyHostToDevice);

	if (err_u != hipSuccess ||
	    err_v != hipSuccess ||
	    err_dens != hipSuccess ||
	    err_u_prev != hipSuccess ||
	    err_v_prev != hipSuccess ||
	    err_dens_prev != hipSuccess)
	{
	    fprintf(stderr, "Error al copiar memoria de host a device\n");
	    return;
	}
	static int times = 1;
        static double react_ns_p_cell = 0.0;
        static double vel_ns_p_cell = 0.0;
        static double dens_ns_p_cell = 0.0;
        static float  milliseconds;
        static hipEvent_t start = nullptr, stop = nullptr, current = nullptr, one_second = nullptr;
        if (one_second == nullptr)
        {
            hipEventCreate(&one_second);
            hipEventRecord(one_second);
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventCreate(&current);
        }

        hipEventRecord(start);
        react ( dens_prev, u_prev, v_prev );
        hipEventRecord(stop);
	hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        react_ns_p_cell += (N * N) / (1.0e3 * milliseconds);

        hipEventRecord(start);
        vel_step ( N, u, v, u_prev, v_prev, visc, dt );
        hipEventRecord(stop);
	hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        vel_ns_p_cell += (N * N) / (1.0e3 * milliseconds);

        hipEventRecord(start);
        dens_step ( N, dens, dens_prev, u, v, diff, dt );
        hipEventRecord(stop);
	hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        dens_ns_p_cell += (N * N) / (1.0e3 * milliseconds);

        hipEventRecord(current);
	hipEventSynchronize(current);
        hipError_t salida = hipEventElapsedTime(&milliseconds, one_second, current);
        if (salida == hipErrorInvalidValue)
	{
		printf("Error al obtener tiempo entre current y one second\n");
		return;
	}
	if (1000.0f <= milliseconds) { /* at least 1s between stats */
                printf("%lf, %lf, %lf, %lf: ns per cell total, react, vel_step, dens_step\n",
                        (react_ns_p_cell+vel_ns_p_cell+dens_ns_p_cell)/times,
                        react_ns_p_cell/times, vel_ns_p_cell/times, dens_ns_p_cell/times);
                hipEventRecord(one_second);
                react_ns_p_cell = 0.0;
                vel_ns_p_cell = 0.0;
                dens_ns_p_cell = 0.0;
                times = 1;
        } else {
                times++;
	}
	hipDeviceSynchronize();
	err_u         = hipMemcpy(hu, u, size, hipMemcpyDeviceToHost);
	err_v         = hipMemcpy(hv, v, size, hipMemcpyDeviceToHost);
        err_dens      = hipMemcpy(hdens, dens, size, hipMemcpyDeviceToHost);
	err_u_prev    = hipMemcpy(hu_prev, u_prev, size, hipMemcpyDeviceToHost);
	err_v_prev    = hipMemcpy(hv_prev, v_prev, size, hipMemcpyDeviceToHost);
	err_dens_prev = hipMemcpy(hdens_prev, dens_prev, size, hipMemcpyDeviceToHost);
	if (err_u != hipSuccess ||
	    err_v != hipSuccess ||
	    err_dens != hipSuccess ||
	    err_u_prev != hipSuccess ||
	    err_v_prev != hipSuccess ||
	    err_dens_prev != hipSuccess)
	{
	    fprintf(stderr, "Error al copiar memoria de device a host\n");
	    return;
	}

		
	glutSetWindow ( win_id );
	glutPostRedisplay ();
}

static void display_func ( void )
{
	pre_display ();

		if ( dvel ) draw_velocity ();
		else		draw_density ();

	post_display ();
}


/*
  ----------------------------------------------------------------------
   open_glut_window --- open a glut compatible window and set callbacks
  ----------------------------------------------------------------------
*/

static void open_glut_window ( void )
{
	glutInitDisplayMode ( GLUT_RGBA | GLUT_DOUBLE );

	glutInitWindowPosition ( 0, 0 );
	glutInitWindowSize ( win_x, win_y );
	win_id = glutCreateWindow ( "Alias | wavefront" );

	glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();

	pre_display ();

	glutKeyboardFunc ( key_func );
	glutMouseFunc ( mouse_func );
	glutMotionFunc ( motion_func );
	glutReshapeFunc ( reshape_func );
	glutIdleFunc ( idle_func );
	glutDisplayFunc ( display_func );
}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main ( int argc, char ** argv )
{
	glutInit ( &argc, argv );

	if ( argc != 1 && argc != 6 ) {
		fprintf ( stderr, "usage : %s N dt diff visc force source\n", argv[0] );
		fprintf ( stderr, "where:\n" );\
		fprintf ( stderr, "\t N      : grid resolution\n" );
		fprintf ( stderr, "\t dt     : time step\n" );
		fprintf ( stderr, "\t diff   : diffusion rate of the density\n" );
		fprintf ( stderr, "\t visc   : viscosity of the fluid\n" );
		fprintf ( stderr, "\t force  : scales the mouse movement that generate a force\n" );
		fprintf ( stderr, "\t source : amount of density that will be deposited\n" );
		exit ( 1 );
	}

#ifndef ND
    N = 128;
#else
    N = ND;
#endif
		dt = 0.1f;
		diff = 0.0f;
		visc = 0.0f;
		force = 5.0f;
		source = 100.0f;
		fprintf ( stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
			N, dt, diff, visc, force, source );

	printf ( "\n\nHow to use this demo:\n\n" );
	printf ( "\t Add densities with the right mouse button\n" );
	printf ( "\t Add velocities with the left mouse button and dragging the mouse\n" );
	printf ( "\t Toggle density/velocity display with the 'v' key\n" );
	printf ( "\t Clear the simulation by pressing the 'c' key\n" );
	printf ( "\t Quit by pressing the 'q' key\n" );

	dvel = 0;

	if ( !allocate_data () ) exit ( 1 );
	clear_data ();

	win_x = 512;
	win_y = 512;
	open_glut_window ();

	glutMainLoop ();

	exit ( 0 );
}
