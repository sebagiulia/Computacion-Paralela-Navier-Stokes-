#include "hip/hip_runtime.h"
/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

    This code is a simple prototype that demonstrates how to use the
    code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
    for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include <stdlib.h>
#include <stdio.h>
#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "solver.h"
#include "timing.h"
#include "indices.h"


/* macros */
#define BLOCK_WIDTH 32
#define IXCUDA(x,y) (rbcuda_idx((x),(y),(n+2)))
#define IX(x,y) (rb_idx((x),(y),(N+2)))

__device__ static inline size_t rbcuda_idx(size_t x, size_t y, size_t dim) {
    size_t base = ((x % 2) ^ (y % 2)) * dim * (dim / 2);
    size_t offset = (x / 2) + y * (dim / 2);
    return base + offset;
}


/* global variables */

static int N;
static float dt, diff, visc;
static float force, source;
static int dvel;

static float * u, * v, * u_prev, * v_prev;
static float * dens, * dens_prev;
static float * hu, * hv, * hu_prev, * hv_prev;
static float * hdens, * hdens_prev;

static int win_id;
static int win_x, win_y;
static int mouse_down[3];
static int omx, omy, mx, my;


/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data ( void )
{
	if ( hu ) free ( hu );
	if ( hv ) free ( hv );
	if ( hu_prev ) free ( hu_prev );
	if ( hv_prev ) free ( hv_prev );
	if ( hdens ) free ( hdens );
	if ( hdens_prev ) free ( hdens_prev );
	if ( u ) hipFree ( u );
	if ( v ) hipFree ( v );
	if ( u_prev ) hipFree ( u_prev );
	if ( v_prev ) hipFree ( v_prev );
	if ( dens ) hipFree ( dens );
	if ( dens_prev ) hipFree ( dens_prev );
}

static void clear_data ( void )
{
	int i, size=(N+2)*(N+2);
	for ( i=0 ; i<size ; i++ ) {
		hu[i] = hv[i] = hu_prev[i] = hv_prev[i] = hdens[i] = hdens_prev[i] = 0.0f;
	}
}

static int allocate_data ( void )
{
	int size = (N+2)*(N+2);

	hipError_t ue = hipMalloc((float **)&u, size*sizeof(float) );
	hipError_t ve = hipMalloc((float **)&v, size*sizeof(float) );
	hipError_t dense = hipMalloc((float **)&dens, size*sizeof(float) );
	hipError_t upreve = hipMalloc((float **)&u_prev, size*sizeof(float) );
	hipError_t vpreve = hipMalloc((float **)&v_prev, size*sizeof(float) );
	hipError_t denspreve = hipMalloc((float **)&dens_prev, size*sizeof(float) );

	if (ue != hipSuccess ||
	    ve != hipSuccess ||
	    dense != hipSuccess ||
	    upreve != hipSuccess ||
	    vpreve != hipSuccess ||
	    denspreve != hipSuccess)
	{
	    fprintf ( stderr, "cannot allocate gpu data\n" );
	    return ( 0 );
	}
	hu_prev	   = (float *) malloc ( size*sizeof(float) );
	hv_prev	   = (float *) malloc ( size*sizeof(float) );
	hdens_prev = (float *) malloc ( size*sizeof(float) );
	hu	   = (float *) malloc ( size*sizeof(float) );
	hv	   = (float *) malloc ( size*sizeof(float) );
	hdens      = (float *) malloc ( size*sizeof(float) );

	if ( !hu || !hv || !hu_prev || !hv_prev || !hdens || !hdens_prev ) {
		fprintf ( stderr, "cannot allocate data\n" );
		return ( 0 );
	}

	return ( 1 );
}


/*
  ----------------------------------------------------------------------
   OpenGL specific drawing routines
  ----------------------------------------------------------------------
*/

static void pre_display ( void )
{
	glViewport ( 0, 0, win_x, win_y );
	glMatrixMode ( GL_PROJECTION );
	glLoadIdentity ();
	gluOrtho2D ( 0.0, 1.0, 0.0, 1.0 );
	glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
}

static void post_display ( void )
{
	glutSwapBuffers ();
}

static void draw_velocity ( void )
{
	int i, j;
	float x, y, h;
 	unsigned int size = (N + 2) * (N + 2);
        hipError_t err_u         = hipMemcpy(hu, u, size * sizeof(float), hipMemcpyDeviceToHost);
	hipError_t err_v         = hipMemcpy(hv, v, size * sizeof(float), hipMemcpyDeviceToHost);
	if (err_u != hipSuccess ||
	    err_v != hipSuccess)
	{
	    fprintf(stderr, "Error al copiar memoria de device a host\n");
	    return;
	}

	h = 1.0f/N;

	glColor3f ( 1.0f, 1.0f, 1.0f );
	glLineWidth ( 1.0f );

	glBegin ( GL_LINES );

		for ( i=1 ; i<=N ; i++ ) {
			x = (i-0.5f)*h;
			for ( j=1 ; j<=N ; j++ ) {
				y = (j-0.5f)*h;

				glVertex2f ( x, y );
				glVertex2f ( x+hu[IX(i,j)], y+hv[IX(i,j)] );
			}
		}

	glEnd ();
}

static void draw_density ( void )
{
	int i, j;
	float x, y, h, d00, d01, d10, d11;
	unsigned int size = (N + 2) * (N + 2);
        hipError_t err_dens      = hipMemcpy(hdens, dens, size * sizeof(float), hipMemcpyDeviceToHost);
	if(err_dens != hipSuccess)
	{
	    fprintf(stderr, "Error al copiar memoria de device a host\n");
	    return;
	}

	h = 1.0f/N;

	glBegin ( GL_QUADS );

		for ( i=0 ; i<=N ; i++ ) {
			x = (i-0.5f)*h;
			for ( j=0 ; j<=N ; j++ ) {
				y = (j-0.5f)*h;

				d00 = hdens[IX(i,j)];
				d01 = hdens[IX(i,j+1)];
				d10 = hdens[IX(i+1,j)];
				d11 = hdens[IX(i+1,j+1)];

				glColor3f ( d00, d00, d00 ); glVertex2f ( x, y );
				glColor3f ( d10, d10, d10 ); glVertex2f ( x+h, y );
				glColor3f ( d11, d11, d11 ); glVertex2f ( x+h, y+h );
				glColor3f ( d01, d01, d01 ); glVertex2f ( x, y+h );
			}
		}

	glEnd ();
}

/*
  ----------------------------------------------------------------------
   relates mouse movements to forces sources
  ----------------------------------------------------------------------
*/

__global__ static void init_u_v_d(unsigned int n, float * d_d, float * d_u, float * d_v, 
                                  float max_velocity2, float max_density, 
                                  float force, float source, 
                                  int mouse_down0, int mouse_down2, 
                                  int mx, int my, int omx, int omy,
                                  int win_x, int win_y) {
    // un solo hilo inicializa u, v, d
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x == 0){
	    if (max_velocity2<0.0000005f) {
		    d_u[IXCUDA(n/2, n/2)] = force * 10.0f;

		    d_v[IXCUDA(n/2, n/2)] = force * 10.0f;
	    }
	    if (max_density<1.0f) {
		    d_d[IXCUDA(n/2, n/2)] = source * 10.0f;
	    }
    
        if ( !mouse_down0 && !mouse_down2 ) return;

	    int i = (int)((       mx /(float)win_x)*n+1);
	    int j = (int)(((win_y-my)/(float)win_y)*n+1);

	    if ( i<1 || i>n || j<1 || j>n ) return;

	    if ( mouse_down0 ) {
		    d_u[IXCUDA(i,j)] = force * (mx-omx);
		    d_v[IXCUDA(i,j)] = force * (omy-my);
	    }

	    if ( mouse_down2 ) {
		    d_d[IXCUDA(i,j)] = source;
	    }
    }
}


__global__ void compute_velocity2(const float* u, const float* v, float* velocity2, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float ui = u[i];
        float vi = v[i];
        velocity2[i] = ui * ui + vi * vi;
    }
}

void update_omx_omy() {
    // modifico omx, omy en el host
    if ( !mouse_down[0] && !mouse_down[2] ) return;
    int i = (int)((       mx /(float)win_x)*N+1);
    int j = (int)(((win_y-my)/(float)win_y)*N+1);
	if ( i<1 || i>N || j<1 || j>N ) return;
    omx = mx;
    omy = my;
}

static void react ( float * d_d, float * d_u, float * d_v )
{
    int size = (N + 2) * (N + 2);
    float max_velocity2, max_density;

    // --- Crear array temporal para velocity^2
    float* d_velocity2;
    hipMalloc(&d_velocity2, size * sizeof(float));

    // --- 1. Calcular u^2 + v^2
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    compute_velocity2<<<numBlocks, blockSize>>>(d_u, d_v, d_velocity2, size);

    // --- 2. Usar CUB para obtener los máximos

    // 2.a max_density
    float* d_out_density;
    void* d_temp_storage_density = nullptr;
    size_t temp_storage_bytes_density = 0;
    hipMalloc(&d_out_density, sizeof(float));
    hipcub::DeviceReduce::Max(nullptr, temp_storage_bytes_density, d_d, d_out_density, size);
    hipMalloc(&d_temp_storage_density, temp_storage_bytes_density);
    hipcub::DeviceReduce::Max(d_temp_storage_density, temp_storage_bytes_density, d_d, d_out_density, size);

    // 2.b max_velocity2
    float* d_out_velocity2;
    void* d_temp_storage_velocity = nullptr;
    size_t temp_storage_bytes_velocity = 0;
    hipMalloc(&d_out_velocity2, sizeof(float));
    hipcub::DeviceReduce::Max(nullptr, temp_storage_bytes_velocity, d_velocity2, d_out_velocity2, size);
    hipMalloc(&d_temp_storage_velocity, temp_storage_bytes_velocity);
    hipcub::DeviceReduce::Max(d_temp_storage_velocity, temp_storage_bytes_velocity, d_velocity2, d_out_velocity2, size);

    // --- 3. Copiar máximos a host
    hipMemcpy(&max_density, d_out_density, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&max_velocity2, d_out_velocity2, sizeof(float), hipMemcpyDeviceToHost);

    // --- 4. Resetear arrays a 0
    hipMemset(d_u, 0, size * sizeof(float));
    hipMemset(d_v, 0, size * sizeof(float));
    hipMemset(d_d, 0, size * sizeof(float));


    init_u_v_d<<<dim3(1), dim3(BLOCK_WIDTH)>>>(N, d_d, d_u, d_v, max_velocity2, max_density, 
                                               force, source, mouse_down[0], mouse_down[2], 
                                               mx, my, omx, omy, win_x, win_y);


    hipDeviceSynchronize();
    update_omx_omy();

    // --- Cleanup
    hipFree(d_velocity2);
    hipFree(d_out_density);
    hipFree(d_out_velocity2);
    hipFree(d_temp_storage_density);
    hipFree(d_temp_storage_velocity);
}


/*
  ----------------------------------------------------------------------
   GLUT callback routines
  ----------------------------------------------------------------------
*/

static void key_func ( unsigned char key, int x, int y )
{
	switch ( key )
	{
		case 'c':
		case 'C':
			clear_data ();
			break;

		case 'q':
		case 'Q':
			free_data ();
			exit ( 0 );
			break;

		case 'v':
		case 'V':
			dvel = !dvel;
			break;
	}
}

static void mouse_func ( int button, int state, int x, int y )
{
	omx = mx = x;
	omy = my = y;

	mouse_down[button] = state == GLUT_DOWN;
}

static void motion_func ( int x, int y )
{
	mx = x;
	my = y;
}

static void reshape_func ( int width, int height )
{
	glutSetWindow ( win_id );
	glutReshapeWindow ( width, height );

	win_x = width;
	win_y = height;
}

static void idle_func ( void )
{
static int size = (N+2)*(N+2);
	static int times = 1;
        static double react_ns_p_cell = 0.0;
        static double vel_ns_p_cell = 0.0;
        static double dens_ns_p_cell = 0.0;
        static float  milliseconds;
        static hipEvent_t start = nullptr, stop = nullptr, current = nullptr, one_second = nullptr;
        if (one_second == nullptr)
        {
            hipEventCreate(&one_second);
            hipEventRecord(one_second);
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventCreate(&current);
        }

        hipEventRecord(start);
        react ( dens_prev, u_prev, v_prev );
        hipEventRecord(stop);
	hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        react_ns_p_cell += (N * N) / (1.0e3 * milliseconds);

        hipEventRecord(start);
        vel_step ( N, u, v, u_prev, v_prev, visc, dt );
        hipEventRecord(stop);
	hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        vel_ns_p_cell += (N * N) / (1.0e3 * milliseconds);

        hipEventRecord(start);
        dens_step ( N, dens, dens_prev, u, v, diff, dt );
        hipEventRecord(stop);
	hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        dens_ns_p_cell += (N * N) / (1.0e3 * milliseconds);

        hipEventRecord(current);
	hipEventSynchronize(current);
        hipError_t salida = hipEventElapsedTime(&milliseconds, one_second, current);
        if (salida == hipErrorInvalidValue)
	{
		printf("Error al obtener tiempo entre current y one second\n");
		return;
	}
	if (1000.0f <= milliseconds) { /* at least 1s between stats */
                printf("%lf, %lf, %lf, %lf: ns per cell total, react, vel_step, dens_step\n",
                        (react_ns_p_cell+vel_ns_p_cell+dens_ns_p_cell)/times,
                        react_ns_p_cell/times, vel_ns_p_cell/times, dens_ns_p_cell/times);
                hipEventRecord(one_second);
                react_ns_p_cell = 0.0;
                vel_ns_p_cell = 0.0;
                dens_ns_p_cell = 0.0;
                times = 1;
        } else {
                times++;
	}
	hipDeviceSynchronize();
	
		
	glutSetWindow ( win_id );
	glutPostRedisplay ();
}

static void display_func ( void )
{
	pre_display ();

		if ( dvel ) draw_velocity ();
		else		draw_density ();

	post_display ();
}


/*
  ----------------------------------------------------------------------
   open_glut_window --- open a glut compatible window and set callbacks
  ----------------------------------------------------------------------
*/

static void open_glut_window ( void )
{
	glutInitDisplayMode ( GLUT_RGBA | GLUT_DOUBLE );

	glutInitWindowPosition ( 0, 0 );
	glutInitWindowSize ( win_x, win_y );
	win_id = glutCreateWindow ( "Alias | wavefront" );

	glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();

	pre_display ();

	glutKeyboardFunc ( key_func );
	glutMouseFunc ( mouse_func );
	glutMotionFunc ( motion_func );
	glutReshapeFunc ( reshape_func );
	glutIdleFunc ( idle_func );
	glutDisplayFunc ( display_func );
}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main ( int argc, char ** argv )
{
	glutInit ( &argc, argv );

	if ( argc != 1 && argc != 6 ) {
		fprintf ( stderr, "usage : %s N dt diff visc force source\n", argv[0] );
		fprintf ( stderr, "where:\n" );\
		fprintf ( stderr, "\t N      : grid resolution\n" );
		fprintf ( stderr, "\t dt     : time step\n" );
		fprintf ( stderr, "\t diff   : diffusion rate of the density\n" );
		fprintf ( stderr, "\t visc   : viscosity of the fluid\n" );
		fprintf ( stderr, "\t force  : scales the mouse movement that generate a force\n" );
		fprintf ( stderr, "\t source : amount of density that will be deposited\n" );
		exit ( 1 );
	}

#ifndef ND
    N = 128;
#else
    N = ND;
#endif
		dt = 0.1f;
		diff = 0.0f;
		visc = 0.0f;
		force = 5.0f;
		source = 100.0f;
		fprintf ( stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
			N, dt, diff, visc, force, source );

	printf ( "\n\nHow to use this demo:\n\n" );
	printf ( "\t Add densities with the right mouse button\n" );
	printf ( "\t Add velocities with the left mouse button and dragging the mouse\n" );
	printf ( "\t Toggle density/velocity display with the 'v' key\n" );
	printf ( "\t Clear the simulation by pressing the 'c' key\n" );
	printf ( "\t Quit by pressing the 'q' key\n" );

	dvel = 0;

	if ( !allocate_data () ) exit ( 1 );
	clear_data ();

	win_x = 512;
	win_y = 512;
	open_glut_window ();

	glutMainLoop ();

	exit ( 0 );
}
