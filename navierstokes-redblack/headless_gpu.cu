#include "hip/hip_runtime.h"
/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

	This code is a simple prototype that demonstrates how to use the
	code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
	for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include <stdlib.h>
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "indices.h"
#include "solver.h"
#include "timing.h"

/* macros */

#define IX(x,y) (rb_idx((x),(y),(N+2)))

/* global variables */

static int N;
static float dt, diff, visc;
static float force, source;

static float * u, * v, * u_prev, * v_prev;
static float * dens, * dens_prev;

static float * hu, * hv, * hu_prev, * hv_prev;
static float * hdens, * hdens_prev;

/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data ( void )
{
	if ( hu ) free ( hu );
	if ( hv ) free ( hv );
	if ( hu_prev ) free ( hu_prev );
	if ( hv_prev ) free ( hv_prev );
	if ( hdens ) free ( hdens );
	if ( hdens_prev ) free ( hdens_prev );
	if ( u ) hipFree ( u );
	if ( v ) hipFree ( v );
	if ( u_prev ) hipFree ( u_prev );
	if ( v_prev ) hipFree ( v_prev );
	if ( dens ) hipFree ( dens );
	if ( dens_prev ) hipFree ( dens_prev );
}

static void clear_data ( void )
{
	int i, size=(N+2)*(N+2);

	for ( i=0 ; i<size ; i++ ) {
		hu[i] = hv[i] = hu_prev[i] = hv_prev[i] = hdens[i] = hdens_prev[i] = 0.0f;
	}
}

static int allocate_data ( void )
{
	int size = (N+2)*(N+2);

	hipError_t ue = hipMalloc((float **)&u, size*sizeof(float) );
	hipError_t ve = hipMalloc((float **)&v, size*sizeof(float) );
	hipError_t dense = hipMalloc((float **)&dens, size*sizeof(float) );
	hipError_t upreve = hipMalloc((float **)&u_prev, size*sizeof(float) );
	hipError_t vpreve = hipMalloc((float **)&v_prev, size*sizeof(float) );
	hipError_t denspreve = hipMalloc((float **)&dens_prev, size*sizeof(float) );

	if (ue != hipSuccess ||
	    ve != hipSuccess ||
	    dense != hipSuccess ||
	    upreve != hipSuccess ||
	    vpreve != hipSuccess ||
	    denspreve != hipSuccess)
	{
	    fprintf ( stderr, "cannot allocate gpu data\n" );
	    return ( 0 );
	}
	hu_prev	   = (float *) malloc ( size*sizeof(float) );
	hv_prev	   = (float *) malloc ( size*sizeof(float) );
	hdens_prev = (float *) malloc ( size*sizeof(float) );
	hu	   = (float *) malloc ( size*sizeof(float) );
	hv	   = (float *) malloc ( size*sizeof(float) );
	hdens      = (float *) malloc ( size*sizeof(float) );

	if ( !hu || !hv || !hu_prev || !hv_prev || !hdens || !hdens_prev ) {
		fprintf ( stderr, "cannot allocate data\n" );
		return ( 0 );
	}

	return ( 1 );
}

__global__ void compute_velocity2(const float* u, const float* v, float* velocity2, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float ui = u[i];
        float vi = v[i];
        velocity2[i] = ui * ui + vi * vi;
    }
}


static void react ( float * d_d, float * d_u, float * d_v )
{
    int size = (N + 2) * (N + 2);
    float max_velocity2, max_density;

    // --- Crear array temporal para velocity^2
    float* d_velocity2;
    hipMalloc(&d_velocity2, size * sizeof(float));

    // --- 1. Calcular u^2 + v^2
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    compute_velocity2<<<numBlocks, blockSize>>>(d_u, d_v, d_velocity2, size);

    // --- 2. Usar CUB para obtener los máximos

    // 2.a max_density
    float* d_out_density;
    void* d_temp_storage_density = nullptr;
    size_t temp_storage_bytes_density = 0;
    hipMalloc(&d_out_density, sizeof(float));
    hipcub::DeviceReduce::Max(nullptr, temp_storage_bytes_density, d_d, d_out_density, size);
    hipMalloc(&d_temp_storage_density, temp_storage_bytes_density);
    hipcub::DeviceReduce::Max(d_temp_storage_density, temp_storage_bytes_density, d_d, d_out_density, size);

    // 2.b max_velocity2
    float* d_out_velocity2;
    void* d_temp_storage_velocity = nullptr;
    size_t temp_storage_bytes_velocity = 0;
    hipMalloc(&d_out_velocity2, sizeof(float));
    hipcub::DeviceReduce::Max(nullptr, temp_storage_bytes_velocity, d_velocity2, d_out_velocity2, size);
    hipMalloc(&d_temp_storage_velocity, temp_storage_bytes_velocity);
    hipcub::DeviceReduce::Max(d_temp_storage_velocity, temp_storage_bytes_velocity, d_velocity2, d_out_velocity2, size);

    // --- 3. Copiar máximos a host
    hipMemcpy(&max_density, d_out_density, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&max_velocity2, d_out_velocity2, sizeof(float), hipMemcpyDeviceToHost);

    // --- 4. Resetear arrays a 0
    hipMemset(d_u, 0, size * sizeof(float));
    hipMemset(d_v, 0, size * sizeof(float));
    hipMemset(d_d, 0, size * sizeof(float));

    // --- 5. Aplicar condiciones si hace falta
    int center = IX(N/2, N/2); 
    if (max_velocity2 < 0.0000005f) {
        float force_val = force * 10.0f;
        hipMemcpy(&d_u[center], &force_val, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&d_v[center], &force_val, sizeof(float), hipMemcpyHostToDevice);
    }
    if (max_density < 1.0f) {
        float source_val = source * 10.0f;
        hipMemcpy(&d_d[center], &source_val, sizeof(float), hipMemcpyHostToDevice);
    }

    // --- Cleanup
    hipFree(d_velocity2);
    hipFree(d_out_density);
    hipFree(d_out_velocity2);
    hipFree(d_temp_storage_density);
    hipFree(d_temp_storage_velocity);
}

static void one_step ( void )
{
	static int times = 1;
	static double start_t = 0.0;
	static double one_second = 0.0;
	static double react_ns_p_cell = 0.0;
	static double vel_ns_p_cell = 0.0;
	static double dens_ns_p_cell = 0.0;
	static int size = (N+2)*(N+2);
	hipError_t err_u         = hipMemcpy(u, hu, size, hipMemcpyHostToDevice);
	hipError_t err_v         = hipMemcpy(v, hv, size, hipMemcpyHostToDevice);
	hipError_t err_dens      = hipMemcpy(dens, hdens, size, hipMemcpyHostToDevice);
	hipError_t err_u_prev    = hipMemcpy(u_prev, hu_prev, size, hipMemcpyHostToDevice);
	hipError_t err_v_prev    = hipMemcpy(v_prev, hv_prev, size, hipMemcpyHostToDevice);
	hipError_t err_dens_prev = hipMemcpy(dens_prev, hdens_prev, size, hipMemcpyHostToDevice);

	if (err_u != hipSuccess ||
	    err_v != hipSuccess ||
	    err_dens != hipSuccess ||
	    err_u_prev != hipSuccess ||
	    err_v_prev != hipSuccess ||
	    err_dens_prev != hipSuccess)
	{
	    fprintf(stderr, "Error al copiar memoria de host a device");
	    return;
	}

	start_t = wtime();
	react ( dens_prev, u_prev, v_prev );
	react_ns_p_cell += (N * N) / (1.0e6 * (wtime() - start_t));

	start_t = wtime();
	vel_step ( N, u, v, u_prev, v_prev, visc, dt );
	vel_ns_p_cell += (N * N) / (1.0e6 * (wtime() - start_t));

	start_t = wtime();
	dens_step ( N, dens, dens_prev, u, v, diff, dt );
	dens_ns_p_cell += (N * N) / (1.0e6 * (wtime() - start_t));

	if (1.0<wtime()-one_second) { /* at least 1s between stats */
		printf("%lf, %lf, %lf, %lf: ns per cell total, react, vel_step, dens_step\n",
			(react_ns_p_cell+vel_ns_p_cell+dens_ns_p_cell)/times,
			react_ns_p_cell/times, vel_ns_p_cell/times, dens_ns_p_cell/times);
		one_second = wtime();
		react_ns_p_cell = 0.0;
		vel_ns_p_cell = 0.0;
		dens_ns_p_cell = 0.0;
		times = 1;
	} else {
		times++;
	}

}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main ( int argc, char** argv)
{
    int i = 0;

    if (argc != 1 && argc != 7) {
        fprintf(stderr, "usage : %s N dt diff visc force source\n", argv[0]);
        fprintf(stderr, "where:\n");
        fprintf(stderr, "\t N      : grid resolution\n");
        fprintf(stderr, "\t dt     : time step\n");
        fprintf(stderr, "\t diff   : diffusion rate of the density\n");
        fprintf(stderr, "\t visc   : viscosity of the fluid\n");
        fprintf(stderr, "\t force  : scales the mouse movement that generate a force\n");
        fprintf(stderr, "\t source : amount of density that will be deposited\n");
        exit(1);
    }

#ifndef ND
    N = 128;
#else
    N = ND;
#endif
    dt = 0.1f;
    diff = 0.0f;
    visc = 0.0f;
    force = 5.0f;
    source = 100.0f;
    fprintf(stderr, "Using: N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
		N, dt, diff, visc, force, source);

    if (!allocate_data()) {
        exit(1);
    }
    clear_data();
    for (i = 0; i < 2048; i++) {
        one_step();
    }
    free_data();

    exit(0);
}
